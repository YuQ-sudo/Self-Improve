#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <memory>

#include ""
#include <stdio.h>
#include <helper_functions.h> // for benchmark purpose

__global__ void reduce_baseline_kernel(float* d_in,float* d_out)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;
    __shared__ float sdata[512];
    //全局内存拷贝到共享内存
    sdata[tid] = d_in[i];
    __syncthreads();
    //一个block内数据进行处理 共享内存数据
    for(int s = 1; s < blockDim.x;s *= 2 )
    {
        if(tid % (2 * s) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // 结果数据
    if (tid == 0) d_out[blockIdx.x] = sdata[0];

}

__global__ void reduce_op1_kernel(float* d_in,float* d_out)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;
    __shared__ float sdata[512];
    //全局内存拷贝到共享内存
    sdata[tid] = d_in[i];
    __syncthreads();

    
    //一个block内数据进行处理 共享内存数据
    for(int s = 1; s < blockDim.x;s *= 2 )
    {
        int index = 2*s*tid;
        //优化过程 相当于减少了一半的wrap操作 相比较于base版本 但存在banck conflict
        if(index < blockDim.x){
            sdata[index]+=sdata[index+s];
        }
        __syncthreads();
    }
    // 结果数据
    if (tid == 0) d_out[blockIdx.x] = sdata[0];

}

__global__ void reduce_op2_kernel(float* d_in,float* d_out)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;
    __shared__ float sdata[512];
    //全局内存拷贝到共享内存
    sdata[tid] = d_in[i];
    __syncthreads();

    //一个block内数据进行处理 共享内存数据
    //消除bank
    //分析：0号线程 + 128线程 1号线程+129线程 第一个wrap 完成  不会 再冲突  新的wrap 32线程 + 160号线程
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            sdata[tid]+=sdata[tid+s];
        }
        __syncthreads();
    }
    
    // 结果数据
    if (tid == 0) d_out[blockIdx.x] = sdata[0];

}


int main()
{
    hipSetDevice(0);

    const int64_t size = 32 * 1024 * 1024;
    float* a = (float *)malloc(size * sizeof(float));
    float* d_a;
    float* d_b;
    hipMalloc((void**)&d_a,size * sizeof(float));

    double cpu_calc = 0;
    for(int64_t i = 0; i < size;i++)
    {
        a[i] = 1;
    }
    for(int64_t i = 0; i < size;i++)
    {
        // if(i % 1024 == 0)
        //     printf("a[%lld]:%0.3f %0.3f\n",i,a[i],cpu_calc);
        cpu_calc +=a[i];
    }

    hipMemcpy(d_a,a,size*sizeof(float),hipMemcpyHostToDevice);

    int block_per_threads = 512;

    int num_blocks = (size - 1) / block_per_threads + 1;

    hipMalloc((void**)&d_b,(num_blocks) * sizeof(float));
    float *res=(float *)malloc((num_blocks)*sizeof(float));

    dim3 grid(num_blocks,1);

    dim3 block(block_per_threads,1);

    StopWatchInterface *timer = NULL;
    float elapsedTimeInMs = 0.0f;
    // float bandwidthInGBs = 0.0f;
    sdkCreateTimer(&timer);
    hipEvent_t start, stop;
    (hipEventCreate(&start));
    (hipEventCreate(&stop));
    int MEMCOPY_ITERATIONS = 1;
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
      sdkStartTimer(&timer);
      reduce_baseline_kernel<<<grid,block>>>(d_a,d_b);

      
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      printf("time cost base:%0.3f ms\n",elapsedTimeInMs);
      sdkResetTimer(&timer);

      sdkStartTimer(&timer);
      reduce_op1_kernel<<<grid,block>>>(d_a,d_b);
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      printf("time cost op1:%0.3f ms\n",elapsedTimeInMs);
      sdkResetTimer(&timer);

      sdkStartTimer(&timer);
      reduce_op2_kernel<<<grid,block>>>(d_a,d_b);
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      printf("time cost op2:%0.3f ms\n",elapsedTimeInMs);
      sdkResetTimer(&timer);
    //   memset(flush_buf, i, FLUSH_SIZE);
    }
    hipMemcpy( res, d_b, (num_blocks)*sizeof(float), hipMemcpyDeviceToHost);


    for(int i = 1; i < num_blocks;i++)
    {
        res[0] += res[i];
    }
    printf("%0.3f %0.3f %0.3f\n",cpu_calc,res[0],res[num_blocks-1]);
    // double time_s = elapsedTimeInMs / 1e3;
    // bandwidthInGBs = (size * float(MEMCOPY_ITERATIONS)) / (double)1e9;
    // bandwidthInGBs = bandwidthInGBs / time_s;
    // printf("bandth:%0.3f GBs\n",bandwidthInGBs);



}




