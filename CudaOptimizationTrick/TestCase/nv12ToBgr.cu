#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "iostream"

void readNv12File(unsigned char* gpu_ptr,std::string src_file)
{
    std::ifstream nv12file(src_file.c_str(),std::ifstream::in | std::ios::binary);
    if(!nv12file.is_open()) return -1;
    int ctx_pitch = 1920;
    int width = 1920;
    int height = 1080;
    int ctx_alignment = 32;//线程束 32个线程
    int batch = 1;
    ctx_pitch += (ctx_pitch % ctx_alignment != 0)
                            ? (ctx_alignment - ctx_pitch % ctx_alignment)
                            : 0; //

    int ctx_heights = std::ceil(height * 3.0f / 2.0f);
    int length = ctx_pitch * ctx_heights;

    unsigned char* cpuNv12Data = (unsigned char *)malloc(length);
    nv12file.read((char *)cpuNv12Data,length);

    hipMemcpy(void*(gpu_ptr),cpuNv12Data,length,hipMemcpyHostToDevice);
    nv12file.close();
}

__global__ void nv12_bgr_kernel()
{



}

void nv12ToBgr(uint8_t *pNv12, int nNv12Pitch, float *pBgr,
    int nRgbPitch, int nWidth, int nHeight,
    int nBatchSize, hipStream_t stream)
{
    
    //block size
    dim3 threads(32,16); 
    dim3 blocks((nWidth / 4 -1)/threads.x + 1,(nHeight / 2 - 1) / threads.y + 1, 1);

    nv12_bgr_kernel<<<blocks,threads,0>>>();


}
int main()
{
    hipSetDevice(0);
    unsigned char *input_nv12;
    hipMalloc((void **)&input_nv12,(1920*1080*3/2));
    readNv12File(input_nv12,"./test1920x1080.nv12");


}