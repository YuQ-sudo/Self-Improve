#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

__global__ void vec_gpu_add(const float* a,const float* b,float* c,int size)
{
    int i = threadIdx.x +  blockDim.x * blockIdx.x;
    if(i < size)
    {
        c[i] = a[i] + b[i];
    }
};

int main()
{
    hipSetDevice(0);
    int size_test = 5000;
    float *a = (float*)malloc(size_test * sizeof(float));
    float *b = (float*)malloc(size_test * sizeof(float));
    float *c = (float*)malloc(size_test * sizeof(float));


    float *c_host = (float*)malloc(size_test * sizeof(float));



    float *device_c;
    hipMalloc((void**)&device_c,size_test * sizeof(float));

    for(int i = 0;i<size_test;i++)
    {
        a[i] = rand() % size_test;
        b[i] = rand() % size_test;
        c[i] = a[i] + b[i];
    }

    float *device_a;
    hipMalloc((void**)&device_a,size_test * sizeof(float));
    hipMemcpy(device_a,a,size_test * sizeof(float),hipMemcpyHostToDevice);

    float *device_b;
    hipMalloc((void**)&device_b,size_test * sizeof(float));
    hipMemcpy(device_b,b,size_test * sizeof(float),hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size_test + threadsPerBlock - 1) / threadsPerBlock;

    vec_gpu_add<<<blocksPerGrid,threadsPerBlock>>>(device_a,device_b,device_c,size_test);

    hipMemcpy(c_host,device_c,size_test * sizeof(float),hipMemcpyDeviceToHost);

    for(int i = 0;i<size_test;i++)
    {
        if(c[i] != c_host[i])
        {
            std::cout<<"error"<<std::endl;
            break;
        }
        else
        {
            printf("%d=======%0.3f=============ok\n",i,c[i]);
        }
    }
    return 0;
}